#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cudaKernel.h"
#include "thrust/device_ptr.h"
#include "thrust/remove.h"
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include "ConstDefine.h"


#define CUDA_CALL(x) { const hipError_t a = (x); if (a!= hipSuccess) { printf("\nCUDA Error: %s(err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}

/*
���м���1�����ģdp
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
iter: �ڼ���dp��λ��outputIdx����������ȫ���ڴ�λ�ã�tra1��tra2�������켣����ǰ�����빲���ڴ棻
*/
//__global__ void DPforward(const int iter, const int* outputIdx,const SPoint *tra1,const SPoint *tra2) {
//	SPoint p1 = tra1[threadIdx.x];
//	SPoint p2 = tra2[iter - threadIdx.x - 1]; //�������ڴ��Ǿۼ����ʵ���
//	bool subcost;
//	if((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
//		subcost = 0;
//	}
//	else
//		subcost = 1;
//
//}

/*
SPoint�汾
case1���켣����С��512
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTra[],candidateTra[][]:�켣
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/
__global__ void EDRDistance_1(SPoint *queryTra, SPoint **candidateTra, int candidateNum, int queryLength, int *candidateLength, int** stateTableGPU, int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= candidateNum) return;
	if ((threadID >= candidateLength[blockID]) && (threadID >= queryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = queryLength + lenT - 1;
	__shared__ short state[2][MAXTHREAD]; //���ڴ洢ǰ���εĽ��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	__shared__ SPoint queryTraS[MAXTHREAD];
	__shared__ SPoint traData[MAXTHREAD];
	if (threadID < lenT) {
		traData[threadID] = candidateTra[blockID][threadID];
	}
	if (threadID < queryLength) {
		queryTraS[threadID] = queryTra[threadID];
	}
	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= queryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = queryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = queryLength;
	}

	int myState;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			if (threadID <= i) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				//if((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
				//	subcost = 0;
				//}
				//else
				//	subcost = 1;
				subcost = !((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON));
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
				//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
				//myState = (state_ismatch < state_up) * state_ismatch + (state_left < state_up) * state_up + (state_left >= state_up) * state_left;

			}
		}
		else if (i > iterNum - len1) {
			if (threadID <= iterNum - i - 1) {
				SPoint p1 = tra1[threadID + len1 - (iterNum - i)];
				SPoint p2 = tra2[len2 - 1 - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if ((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID + 1] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
			}
		}
		else
		{
			if (threadID < len1) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if ((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
			}
		}
		//дmyState��share�ڴ�,ckecked
		int startidx;
		//���Ƚ�������д��ȫ���ڴ棬ȫд
		//startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		//����Ӧд��ȫ���ڴ����ʼλ��

		if (i - 2 < len1 - 2) {
			startidx = (i - 2 + 2)*(i - 2 + 3) / 2;
			if (threadID <= i) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else if (i - 2 >= iterNum - len1) {
			startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i - 2))*(iterNum - (i - 2) + 1) / 2;
			if (threadID <= iterNum - i + 1) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else
		{
			startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
			if (threadID <= len1) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}

		//�ƶ������ݵ�������
		state[0][threadID] = state[1][threadID];
		//д��������
		if (i < len1 - 1) {
			if (threadID <= i)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			if (threadID <= iterNum - i - 1)
				state[1][threadID] = myState;
		}
		else
		{
			if (threadID < len1)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
		__syncthreads();
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0)
		result[blockID] = myState;
}


//__global__ void testSharedMemory()
//{
//	__shared__ SPoint queryTraS[MAXLENGTH];
//	__shared__ SPoint traData[MAXLENGTH];
//	__shared__ SPoint traData2[MAXLENGTH];
//	SPoint s;
//	s.x = 4;
//	s.y = 5;
//	traData[1535] = s;
//	queryTraS[1535] = s;
//	traData2[1535] = s;
//}

/*
SPoint�汾
ͬʱ�������ɸ�query��EDR��������һ��EDR����Ϊ��λ��ÿ��block����һ��EDR��thread����һ��б����state�Ĳ��м��㡣
case1���켣���ȿɳ���512������ѭ���������512�ġ�
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTaskNum:�ܹ��м���EDR��������
queryTaskInfo[]��ÿ��task��Ӧ��qID��candidateID��Ϣ����struct�洢
queryTra[],candidateTra[]:�켣���ݣ�candidateTra��֤���ڲ��켣���ظ�
queryTraOffset[],candidateTraOffset[]:ÿ���켣��offset��candidateTra��֤���ڲ��켣���ظ�
queryLength[],candidateLength[]:ÿ���켣�ĳ��ȣ���ʵoffset������ǳ��ȣ�����idx������Ķ�Ӧ
����candidateLength[id]�ǵ�id��candidate Traj�ĳ���
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/

// EDRDistance_Batch_Handler(validCandTrajNum, taskInfoTableGPU, queryTraGPUBase, queryTraOffsetGPU, candidateOffsetsGPU, queryLengthGPU, candidateTraLengthGPU, resultReturnedGPU, &defaultStream);
__global__ void EDRDistance_Batch(int queryTaskNum, TaskInfoTableForSimilarity* taskInfoTable, SPoint *queryTra, int* queryTraOffset, SPoint** candidateTraOffsets, int* queryLength, int *candidateLength, int *result) {

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;

	if (blockID >= queryTaskNum) return;

	__shared__ int thisQueryID;
	__shared__ int thisQueryLength; 
	__shared__ int lenT;
	__shared__ int iterNum;

	thisQueryID = taskInfoTable[blockID].qID;
	if(thisQueryID < 0 ) return; // block���ֿ���

	__shared__ short state[2][MAXLENGTH + 1];

	__shared__ SPoint *queryTraS;
	__shared__ SPoint *traData;
	__shared__ SPoint *tra1, *tra2;
	__shared__ int len1, len2;

	if (threadID == 0) {
		thisQueryID = taskInfoTable[blockID].qID;
		thisQueryLength = queryLength[thisQueryID];
		lenT = candidateLength[blockID];
		iterNum = thisQueryLength + lenT - 1;
		state[0][0] = 0;
		// state[0][1] = 1;
		state[1][0] = 1;
		state[1][1] = 1;
		queryTraS = queryTra + queryTraOffset[thisQueryID]; 
		traData = candidateTraOffsets[blockID]; 
		if (lenT >= thisQueryLength) {
			tra1 = queryTraS;
			tra2 = traData;
			len1 = thisQueryLength;
			len2 = lenT;
		}
		else
		{
			tra1 = traData;
			tra2 = queryTraS;
			len1 = lenT;
			len2 = thisQueryLength;
		}
	}

	__syncthreads(); // ͬ����֤�����߳̿���

	if ((threadID >= lenT) && (threadID >= thisQueryLength)) return;	
	//__shared__ SPoint queryTraS[MAXLENGTH];
	//__shared__ SPoint traData[MAXLENGTH];

	//for (int i = 0; i <= lenT - 1;i+=MAXTHREAD)
	//{
	//	if(threadID+i<lenT)
	//	{
	//		traData[threadID + i] = SPoint(candidateTraOffsets[blockID][threadID + i]);
	//	}
	//}

	//SPoint* queryTraBaseAddr = queryTra + queryTraOffset[thisQueryID];
	//for (int i = 0; i <= thisQueryLength - 1;i+=MAXTHREAD)
	//{
	//	if(threadID+i<thisQueryLength)
	//	{
	//		queryTraS[threadID + i] = *(queryTraBaseAddr + threadID + i);
	//	}
	//}
	
	int myState[5];// 256*4 = 1024 �� __shared ����
	int nodeID;
	SPoint p1;
	SPoint p2;
	bool subcost;

	for (int i = 0; i <= iterNum - 1; i++) { // block ���ѭ��
		if (i < len1 - 1) {
			for (int startIdx = 0; startIdx <= i; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID; 
				if (nodeID <= i) {

					p1 = tra1[nodeID]; // fetch from global memory
					p2 = tra2[i - nodeID]; // fetch from global memory

					// id1 + id2 = i
					if ((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
						subcost = 0;
					}
					else
						subcost = 1;
					//subcost = !((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON));
					bool c1 = ((state[0][nodeID] + subcost < (state[1][nodeID] + 1)) && (state[0][nodeID] + subcost < (state[1][nodeID + 1] + 1)));
					bool c2 = (((state[1][nodeID + 1] + 1) < (state[1][nodeID] + 1)) && (((state[1][nodeID + 1] + 1) < state[0][nodeID] + subcost)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * (state[0][nodeID] + subcost) + c2 * (state[1][nodeID + 1] + 1) + !(c1 || c2) * (state[1][nodeID] + 1);
					//if ((state_ismatch < state_up) && (state_ismatch < state_left))
					//	myState[nodeID/MAXTHREAD] = state_ismatch;
					//else if ((state_left < state_up) && ((state_left < state_ismatch)))
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					////ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					//myState[nodeID / MAXTHREAD] = (state_ismatch < state_up) && (state_ismatch < state_left) * state_ismatch + ((state_left < state_up) && ((state_left < state_ismatch))) * state_left + !(((state_ismatch < state_up) && (state_ismatch < state_left))||(((state_left < state_up) && ((state_left < state_ismatch))))) * state_up;
				}
			}
		}
		else if (i > iterNum - len1) {
			for (int startIdx = 0; startIdx <= iterNum - i - 1; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= iterNum - i - 1) {
					// EDR�����������
					p1 = tra1[nodeID + len1 - (iterNum - i)]; // fetch from global memory
					p2 = tra2[len2 - 1 - nodeID];
					if ((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
						subcost = 0;
					}
					else
						subcost = 1;
					//if (state_ismatch < state_up)
					//	myState[nodeID / MAXTHREAD] = state_ismatch;
					//else if (state_left < state_up)
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					bool c1 = (((state[0][nodeID + 1] + subcost) < (state[1][nodeID] + 1)) && ((state[0][nodeID + 1] + subcost) < (state[1][nodeID + 1] + 1)));
					bool c2 = (((state[1][nodeID + 1] + 1) < (state[1][nodeID] + 1)) && (((state[1][nodeID + 1] + 1) < (state[0][nodeID + 1] + subcost))));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * (state[0][nodeID + 1] + subcost) + c2 * (state[1][nodeID + 1] + 1) + !(c1 || c2) * (state[1][nodeID] + 1);
				}
			}
		}
		else
		{
			for (int startIdx = 0; startIdx < len1; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID < len1) { // ע�������ж� ��֤������̸߳������ø��Ǽ��� �����߳̿���
					p1 = tra1[nodeID]; // fetch from global memory
					p2 = tra2[i - nodeID]; //�������ڴ��Ǿۼ����ʵ���?
					if ((fabs(p1.x - p2.x) < EPSILON) && (fabs(p1.y - p2.y)<EPSILON)) {
						subcost = 0;
					}
					else
						subcost = 1;
					//int state_ismatch = (state[0][nodeID] + subcost);
					//int state_up = (state[1][nodeID] + 1);
					//int state_left = (state[1][nodeID + 1] + 1);
					//if (state_ismatch < state_up)
					//	myState[nodeID / MAXTHREAD] = state_ismatch;
					//else if (state_left < state_up)
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					bool c1 = (((state[0][nodeID] + subcost) < (state[1][nodeID] + 1)) && ((state[0][nodeID] + subcost) < (state[1][nodeID + 1] + 1)));
					bool c2 = (((state[1][nodeID + 1] + 1) < (state[1][nodeID] + 1)) && (((state[1][nodeID + 1] + 1) < (state[0][nodeID] + subcost))));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * (state[0][nodeID] + subcost) + c2 * (state[1][nodeID + 1] + 1) + !(c1 || c2) * (state[1][nodeID] + 1);
				}
			}
		}

		// ����state[0]
		//state[1] �� state[0]
		for (int Idx = 0; Idx < MAXLENGTH; Idx += MAXTHREAD)
		{
			if(threadID + Idx < MAXLENGTH)
				state[0][threadID + Idx] = state[1][threadID + Idx];
		}
		//state[0][threadID] = state[1][threadID];

		// ����state[1]
		//д��������
		if (i < len1 - 1) {
			for (int Idx = 0; Idx <= i; Idx += MAXTHREAD) {
				if (threadID + Idx <= i)
					state[1][Idx + threadID + 1] = myState[Idx / MAXTHREAD];
			}
			// �����׶�
			if (threadID == 0) {
				state[1][0] = i + 2; // ����ͷ
				state[1][i + 2] = i + 2; // ����β
			}
		}
		else if (i >= iterNum - len1) {
			//if (threadID <= iterNum - i - 1)
			//	state[1][threadID] = myState;
			for (int Idx = 0; Idx <= iterNum - i - 1; Idx += MAXTHREAD) {
				if (threadID + Idx <= iterNum - i - 1)
					state[1][threadID + Idx] = myState[Idx / MAXTHREAD];
			}
			// �����׶β��ø���
		}
		else
		{
			//if (threadID < len1)
			//	state[1][threadID + 1] = myState;
			//if (threadID == 0) {
			//	state[1][0] = i + 2;
			//}
			for (int Idx = 0; Idx <= len1; Idx += MAXTHREAD) {
				if (threadID + Idx < len1)
					state[1][Idx + threadID + 1] = myState[Idx / MAXTHREAD];
			}
			// ���ֽ׶�
			if (threadID == 0) {
				state[1][0] = i + 2; // ֻ�����ͷ 
			}
		}

		__syncthreads(); // ͬ��һ��block��thread

	}
	// kernel Ӧ��ûʲô������
	// for ѭ������ �����EDR��� 
	// myState[0]; ����EDR
	if (threadID == 0 && blockID < queryTaskNum)
		result[blockID] = myState[0];
	// std::cout << "calc EDR success!\n" ;
}
// EDRDistance_Batch_Handler(validCandTrajNum, taskInfoTableGPU, queryTraGPUBase, queryTraOffsetGPU, candidateOffsetsGPU, queryLengthGPU, candidateTraLengthGPU, resultReturnedGPU, &defaultStream);
int EDRDistance_Batch_Handler(int queryTaskNum, TaskInfoTableForSimilarity* taskInfoTable, SPoint *queryTra, int* queryTraOffset, SPoint** candidateTraOffsets, int* queryLength, int *candidateLength, int *result, hipStream_t *stream)
{
	//printf("run kernel now\n");
	EDRDistance_Batch <<< queryTaskNum, MAXTHREAD, 0, *stream >> >(queryTaskNum, taskInfoTable, queryTra, queryTraOffset, candidateTraOffsets, queryLength, candidateLength, result);
	//CUDA_CALL(hipDeviceSynchronize());
	return 0;
}

__device__ inline int binary_search_intPair(intPair* temp, int left, int right, int val)
{
	int mid = (left + right) / 2;
	while (left <= right)
	{
		mid = (left + right) / 2;
		if (temp[mid].int_1 == val)
			return temp[mid].int_2;
		else if (temp[mid].int_1 > val)
		{
			right = mid - 1;
		}
		else
			left = mid + 1;
	}
	return 0;
}

__device__ inline int binary_search_intPair_Neighbor(intPair* temp, int left, int right, int val)
{
	int mid = (left + right) / 2;
	while (left <= right)
	{
		mid = (left + right) / 2;
		if (temp[mid].int_1 == val)
			return mid;
		else if (temp[mid].int_1 > val)
		{
			right = mid - 1;
		}
		else
			left = mid + 1;
	}
	return -1;
}

// -1Ϊû�ҵ�
__device__ inline int binary_search_int(int* temp, int left, int right, int val)
{
	int mid = (left + right) / 2;
	while (left <= right)
	{
		mid = (left + right) / 2;
		if (temp[mid] == val)
			return mid;
		else if (temp[mid] > val)
		{
			right = mid - 1;
		}
		else
			left = mid + 1;
	}
	return -1;
}

__device__ inline int getIdxFromXYGPU(int x, int y)
{
	int lenx, leny;
	if (x == 0)
		lenx = 1;
	else
	{
		lenx = int(log2f(x)) + 1;
	}
	if (y == 0)
		leny = 1;
	else
		leny = int(log2f(y)) + 1;
	int result = 0;
	int xbit = 1, ybit = 1;
	for (int i = 1; i <= 2 * max(lenx, leny); i++)
	{
		if ((i & 1) == 1) //����
		{
			result += (x >> (xbit - 1) & 1) * (1 << (i - 1));
			xbit = xbit + 1;
		}
		else //ż��
		{
			result += (y >> (ybit - 1) & 1) * (1 << (i - 1));
			ybit = ybit + 1;
		}
	}
	return result;
}

__device__ inline int findNeighborGPU(int cellNum, int cellID, int * neighborID)
{
	int x = 0, y = 0;
	for (int bit = 0; bit <= int(log2f(cellNum)) - 1; bit++) {
		if (bit % 2 == 0) {
			//����λ
			x += ((cellID >> bit)&(1))*(1 << (bit / 2));
		}
		else {
			//ż��λ
			y += ((cellID >> bit)&(1))*(1 << (bit / 2));
		}
	}
	int cnt = 0;
	for (int xx = x - 1; xx <= x + 1; xx++) {
		for (int yy = y - 1; yy <= y + 1; yy++) {
			if ((xx != x) || (yy != y))
				neighborID[cnt++] = getIdxFromXYGPU(xx, yy);
			//printf("%d\t", cnt);
		}
	}
	return 0;
}

__device__ inline bool isPositive(short x)
{
	return x >= 0;
}

__global__ void Calculate_FD_Sparse(intPair* queryFVGPU, intPair* FVinfo, intPair* FVTable, intPair* SubbedArray, intPair* SubbedArrayOffset, int SubbedArrayJump, int queryCellLength, int startTrajIdx, int checkNum, int cellNum, int trajNumInDB, int nonZeroFVNumInDB, short* FDistance)
{
	//��һ�׶Σ����м���
	const int MAX_QUERY_CELLNUMBER = 512;
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int threadIDGlobal = blockDim.x*blockID + threadID;

	__shared__ intPair queryCellTraj[MAX_QUERY_CELLNUMBER];
	__shared__ intPair dbCellTraj[MAX_QUERY_CELLNUMBER];
	//cellchecked��¼��query�г��ֵ�cell��ţ������ڷ��������ʱ�����ǲ����Ѿ������ˡ��Ժ�����ڹ鲢���и��ô˱�����
	__shared__ int cellChecked[MAX_QUERY_CELLNUMBER];
	for (int i = 0; i <= queryCellLength - 1; i += MAXTHREAD) {
		if (threadID + i < queryCellLength)
		{
			queryCellTraj[threadID + i] = queryFVGPU[threadID + i];
		}
	}
	int dbTrajStartIdx = FVinfo[startTrajIdx + blockID].int_2;
	int dbTrajEndIdx;
	if (blockID + startTrajIdx == trajNumInDB - 1)
		dbTrajEndIdx = nonZeroFVNumInDB - 1;
	else
		dbTrajEndIdx = FVinfo[startTrajIdx + blockID + 1].int_2 - 1;

	for (int i = 0; i <= dbTrajEndIdx - dbTrajStartIdx; i += MAXTHREAD)
	{
		if (threadID + i <= dbTrajEndIdx - dbTrajStartIdx)
			dbCellTraj[threadID + i] = FVTable[dbTrajStartIdx + threadID + i];
	}
	//1.1:��query��ȥdb
	for (int i = 0; i < queryCellLength; i += MAXTHREAD)
	{
		if (threadID + i < queryCellLength) {
			int find = binary_search_intPair(dbCellTraj, 0, dbTrajEndIdx - dbTrajStartIdx, queryCellTraj[threadID + i].int_1);
			cellChecked[threadID + i] = queryCellTraj[threadID + i].int_1;
			SubbedArray[SubbedArrayJump * blockID + threadID + i].int_1 = queryCellTraj[threadID + i].int_1;
			SubbedArray[SubbedArrayJump * blockID + threadID + i].int_2 = queryCellTraj[threadID + i].int_2 - find;
		}
		if (threadID == 0) {
			SubbedArrayOffset[blockID].int_1 = queryCellLength - 1;
			SubbedArrayOffset[blockID].int_2 = queryCellLength + dbTrajEndIdx - dbTrajStartIdx;
		}
	}
	//1.2����db��ȥquery��ע��Ӹ���
	for (int i = 0; i <= dbTrajEndIdx - dbTrajStartIdx; i += MAXTHREAD)
	{
		if (threadID + i <= dbTrajEndIdx - dbTrajStartIdx)
		{
			intPair cellNo = dbCellTraj[threadID + i];
			int find = binary_search_int(cellChecked, 0, queryCellLength - 1, cellNo.int_1);
			if (find == -1)
			{
				SubbedArray[SubbedArrayJump * blockID + queryCellLength + threadID + i].int_1 = cellNo.int_1;
				SubbedArray[SubbedArrayJump * blockID + queryCellLength + threadID + i].int_2 = -cellNo.int_2;
			}
			else
				SubbedArray[SubbedArrayJump * blockID + queryCellLength + threadID + i].int_1 = -1;
		}
	}
	__syncthreads();
	//�ڶ��׶Σ��������ڣ�������
	//����׶θ�Ϊÿ��thread����һ��FD
	//2.1���ϲ�ÿ��subbedArray
	if (threadIDGlobal < checkNum) {
		int startMergeIdx = SubbedArrayOffset[threadIDGlobal].int_1 + 1;
		int endMergeIdx = SubbedArrayOffset[threadIDGlobal].int_2;
		int frontPtr = startMergeIdx;
		for (int i = startMergeIdx; i <= endMergeIdx; i++)
		{
			if (SubbedArray[SubbedArrayJump * threadIDGlobal + i].int_1 != -1)
			{
				SubbedArray[SubbedArrayJump * threadIDGlobal + frontPtr] = SubbedArray[SubbedArrayJump * threadIDGlobal + i];
				frontPtr++;
			}
		}
		SubbedArrayOffset[threadIDGlobal].int_2 = frontPtr - 1;
	}
	//2.2 ��������
	int neighborsID[8];
	//cell����ָ�ڼ���Ԫ��
	for (int cell = 0; cell <= SubbedArrayOffset[threadIDGlobal].int_2; cell++)
	{
		findNeighborGPU(cellNum, cell, neighborsID);
		//for (int i = 0; i <= 7; i++)
		//	neighborsID[i] = 11;
		for (int i = 0; i <= 7; i++)
		{
			int find = binary_search_intPair_Neighbor(&SubbedArray[SubbedArrayJump * threadIDGlobal], 0, SubbedArrayOffset[threadIDGlobal].int_1, neighborsID[i]);
			if (find == -1) {
				find = binary_search_intPair_Neighbor(&SubbedArray[SubbedArrayJump * threadIDGlobal], SubbedArrayOffset[threadIDGlobal].int_1 + 1, SubbedArrayOffset[threadIDGlobal].int_2, neighborsID[i]);
			}
			// �����-1��˵�����neighbor��0�����ô���
			if (find != -1)
			{
				if (isPositive(SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2) != isPositive(SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2))
				{
					if (fabsf(SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2) > fabsf(SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2))
					{
						SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2 = SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2 + SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2;
						SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2 = 0;
					}
					else {
						SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2 = SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2 + SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2;
						SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2 = 0;
						break;
					}
				}
			}
		}
	}
	__syncthreads();
	//�����׶Σ�ͳ����������
	//��Ȼ��ÿ��block����һ��FD�ļ���
	if (blockID >= checkNum)
		return;
	int *tempsumPosi = (int*)queryCellTraj;
	int *tempsumNega = (int*)dbCellTraj;
	tempsumPosi[threadID] = 0;
	tempsumNega[threadID] = 0;
	for (int i = 0; i <= SubbedArrayOffset[blockID].int_2; i += MAXTHREAD)
	{
		if (i + threadID <= SubbedArrayOffset[blockID].int_2)
		{
			tempsumPosi[threadID] += (isPositive(SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2)*SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2);
			tempsumNega[threadID] += (-(!isPositive(SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2))*SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2);
		}
	}
	__shared__ int sizeOfTempSum;
	if (threadID == 0)
		sizeOfTempSum = MAXTHREAD;
	__syncthreads();
	while ((sizeOfTempSum>1))
	{
		if (threadID <= (sizeOfTempSum >> 1) - 1)
		{
			tempsumPosi[threadID] = tempsumPosi[threadID] + tempsumPosi[threadID + (sizeOfTempSum >> 1)];
			tempsumNega[threadID] = tempsumNega[threadID] + tempsumNega[threadID + (sizeOfTempSum >> 1)];
		}
		__syncthreads();
		if (threadID == 0)
			sizeOfTempSum = (sizeOfTempSum >> 1);
		__syncthreads();
	}
	if (threadID == 0)
		FDistance[blockID] = (tempsumPosi[0] > tempsumNega[0]) ? tempsumPosi[0] : tempsumNega[0];
}

//ÿ��block����һ��FD�ļ���
__global__ void Calculate_FD_NonColumn(short* queryFVGPU, intPair* FVinfo, intPair* FVTable, int startTrajIdx, int checkNum, int cellNum, int trajNumInDB, int nonZeroFVNumInDB, short* FDistance)
{
	//��һ�׶Σ����м���
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int threadIDGlobal = blockDim.x*blockID + threadID;
	if (blockID >= checkNum)
		return;
	__shared__ intPair taskInfo;
	if (threadID == 0)
		taskInfo = FVinfo[blockID + startTrajIdx];
	int nextCnt;
	if (blockID + startTrajIdx == trajNumInDB - 1)
		nextCnt = nonZeroFVNumInDB;
	else
		nextCnt = FVinfo[blockID + startTrajIdx + 1].int_2;
	__syncthreads();
	for (int i = 0; i <= (cellNum - 1); i += MAXTHREAD)
	{
		int find = binary_search_intPair(FVTable, taskInfo.int_2, (nextCnt - 1), (i + threadID));
		//int find = 1;
		//int k = cellNum*blockID + (i + threadID);
		//queryFVGPU[cellNum*blockID + (i + threadID)] = 2;
		queryFVGPU[cellNum*blockID + (i + threadID)] = queryFVGPU[cellNum*blockID + (i + threadID)] - find;
	}
	//�ڶ��׶Σ��������ڣ�������
	//����׶θ�Ϊÿ��thread����һ��FD
	int neighborsID[8];
	for (int cell = 0; cell <= cellNum - 1; cell++)
	{
		//ֻ��Ҫһ�����߳̾�����
		if (threadIDGlobal >= checkNum)
			break;
		if (queryFVGPU[cellNum*threadIDGlobal + cell] != 0)
		{
			findNeighborGPU(cellNum, cell, neighborsID);
			//for (int i = 0; i <= 7; i++)
			//	neighborsID[i] = 11;
			for (int i = 0; i <= 7; i++)
			{
				if (isPositive(queryFVGPU[cellNum*threadIDGlobal + cell]) != isPositive(queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]])) {
					if (fabsf(queryFVGPU[cellNum*threadIDGlobal + cell]) > fabsf(queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]]))
					{
						queryFVGPU[cellNum*threadIDGlobal + cell] = queryFVGPU[cellNum*threadIDGlobal + cell] + queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]];
						queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]] = 0;
					}
					else
					{
						queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]] = queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]] + queryFVGPU[cellNum*threadIDGlobal + cell];
						queryFVGPU[cellNum*threadIDGlobal + cell] = 0;
						break;
					}
				}
			}
		}
	}
	__syncthreads();
	//�����׶Σ�ͳ����������
	//��Ȼ��ÿ��block����һ��FD�ļ���
	__shared__ int tempsumPosi[MAXTHREAD], tempsumNega[MAXTHREAD];
	tempsumPosi[threadID] = 0;
	tempsumNega[threadID] = 0;
	for (int i = 0; i <= cellNum - 1; i += MAXTHREAD)
	{
		tempsumPosi[threadID] += (isPositive(queryFVGPU[blockID*cellNum + (i + threadID)])*queryFVGPU[blockID*cellNum + (i + threadID)]);
		tempsumNega[threadID] += (-(!isPositive(queryFVGPU[blockID*cellNum + (i + threadID)]))*queryFVGPU[blockID*cellNum + (i + threadID)]);
	}
	__shared__ int sizeOfTempSum;
	if (threadID == 0)
		sizeOfTempSum = MAXTHREAD;
	__syncthreads();
	while ((sizeOfTempSum>1))
	{
		if (threadID <= (sizeOfTempSum >> 1) - 1)
		{
			tempsumPosi[threadID] = tempsumPosi[threadID] + tempsumPosi[threadID + (sizeOfTempSum >> 1)];
			tempsumNega[threadID] = tempsumNega[threadID] + tempsumNega[threadID + (sizeOfTempSum >> 1)];
		}
		__syncthreads();
		if (threadID == 0)
			sizeOfTempSum = (sizeOfTempSum >> 1);
		__syncthreads();
	}
	if (threadID == 0)
		FDistance[blockID] = (tempsumPosi[0] > tempsumNega[0]) ? tempsumPosi[0] : tempsumNega[0];

}

//SubbedArrayJump��SubbedArray��ÿһ���ж��ٸ�Ԫ�أ�������idx��
int Similarity_Pruning_Handler(intPair* queryFVGPU, intPair* FVinfo, intPair* FVTable, intPair* SubbedArray, intPair* SubbedArrayOffset, int SubbedArrayJump, int queryCellLength, int startTrajIdx, int checkNum, int cellNum, int trajNumInDB, int nonZeroFVNumInDB, short* FDistance, hipStream_t stream)
{
#ifdef NOT_COLUMN_ORIENTED
	Calculate_FD_NonColumn << <checkNum, MAXTHREAD, 0, stream >> >(queryFVGPU, FVinfo, FVTable, startTrajIdx, checkNum, cellNum, trajNumInDB, nonZeroFVNumInDB, FDistance);
#else
	Calculate_FD_Sparse << <checkNum, MAXTHREAD, 0, stream >> >(queryFVGPU, FVinfo, FVTable, SubbedArray, SubbedArrayOffset, SubbedArrayJump, queryCellLength, startTrajIdx, checkNum, cellNum, trajNumInDB, nonZeroFVNumInDB, FDistance);
#endif
	return 0;
}


/*
//�Ȱ����ܷ���һ��SMִ��һ��DP�����������ٷֱ��������kernel
//constructing...
���Ż���
1��queryTra��queryLength����candidateLength����ͨ����ֵ�ķ�ʽֱ�Ӵ��ݵ�SM�ļĴ���������ȫ���ڴ��ʹ��

*/
int handleEDRdistance(SPoint *queryTra, SPoint **candidateTra, int candidateNum, int queryLength, int *candidateLength, int *result) {
	MyTimer time1;
	time1.start();

	int** stateTableGPU = NULL;
	//��GPU��Ϊ״̬������ڴ�
	int** temp = NULL;
	temp = (int**)malloc(sizeof(int*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&temp[i], sizeof(int)*(candidateLength[i] + 1)*(queryLength + 1)));
	}
	CUDA_CALL(hipMalloc((void***)&stateTableGPU, sizeof(int*)*candidateNum));
	CUDA_CALL(hipMemcpy(stateTableGPU, temp, candidateNum*sizeof(int*), hipMemcpyHostToDevice));

	//Ϊ�洢�Ĺ켣��Ϣ�����ڴ�
	SPoint *queryTraGPU = NULL, **candidateTraGPU = NULL;
	int *candidateLengthGPU = NULL, *resultGPU = NULL;
	CUDA_CALL(hipMalloc((void**)&queryTraGPU, sizeof(SPoint)*queryLength));
	CUDA_CALL(hipMalloc((void**)&candidateLengthGPU, sizeof(int)*candidateNum));
	//CUDA_CALL(hipMalloc((void**)&resultGPU, sizeof(int)*candidateNum));

	SPoint **tempS = (SPoint**)malloc(sizeof(SPoint*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&tempS[i], sizeof(SPoint)*candidateLength[i]));

	}
	CUDA_CALL(hipMalloc((void***)&candidateTraGPU, sizeof(SPoint*)*candidateNum));
	CUDA_CALL(hipMemcpy(candidateTraGPU, tempS, candidateNum*sizeof(SPoint*), hipMemcpyHostToDevice));
	//
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//
	//���ͨ���������ķ������ݹ켣�����Ҫ��켣�����洢
	//��GPU���ݹ켣��Ϣ
	CUDA_CALL(hipMemcpy(queryTraGPU, queryTra, queryLength*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(candidateLengthGPU, candidateLength, candidateNum*sizeof(int), hipMemcpyHostToDevice));

	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMemcpy(tempS[i], candidateTra[i], candidateLength[i] * sizeof(SPoint), hipMemcpyHostToDevice));
	}
	//for (int i = 0; i <= candidateNum - 1;i++)
	//	CUDA_CALL(hipMemcpy(candidateTraGPU[i], candidateTra[i], candidateLength[i]*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipHostAlloc((void**)&result, candidateNum*sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped));
	CUDA_CALL(hipHostGetDevicePointer(&resultGPU, result, 0));
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//ִ��kernel
	EDRDistance_1 << <candidateNum, MAXTHREAD >> >(queryTraGPU, candidateTraGPU, candidateNum, queryLength, candidateLengthGPU, stateTableGPU, resultGPU);

	//ȡ���
	//result = (int*)malloc(candidateNum*sizeof(int));
	//CUDA_CALL(hipMemcpy(result, resultGPU, candidateNum*sizeof(int), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	//	for (int j = 0; j <= candidateNum - 1;j++)
	//		std::cout << result[j] << std::endl;

	//free GPU!!!!!
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	return 0;

}


inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
}


//using namespace thrust;
//static const int MAXTHREAD = 512; //ÿ��block�߳���

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void CUDAwarmUp() {
	CUDA_CALL(hipSetDeviceFlags(hipDeviceMapHost));
	CUDA_CALL(hipSetDevice(0));
	
}

#ifdef _CELL_BASED_STORAGE
int putCellDataSetIntoGPU(Point* pointsPtr, Point*& pointsPtrGPU, int pointNum) {
	
	CUDA_CALL(hipMalloc((void**)&pointsPtrGPU, pointNum * sizeof(Point))); //�������ݵ��ڴ�
	//debug
	//std::cout << pointNum << std::endl;
	//debug
	CUDA_CALL(hipMemcpy(pointsPtrGPU, pointsPtr, pointNum * sizeof(Point), hipMemcpyHostToDevice));//���ݿ�����gpu��
	return 0;
}
__global__ void cudaRangeQuery(int* rangeStarts, int* rangeEnds, int candidateCellNum, const Point* pointsPtr, const float xmin, const float ymin, const float xmax, const float ymax, const int *resultOffset, Point* resultPtrCuda) {
	int cellNo = blockIdx.x; //candidate����ڼ���cell 0,1,2,....
	if (cellNo >= candidateCellNum) return;
	int tid = threadIdx.x;
	if (tid >= 256) return;
	int pointNum = rangeEnds[cellNo] - rangeStarts[cellNo] + 1;//blockҪ��������cell����ô�����
	const int offset = rangeStarts[cellNo];
	for (int i = tid; i <= pointNum - 1; i += MAXTHREAD) {
		float x = pointsPtr[offset + i].x;
		float y = pointsPtr[offset + i].y;
		uint32_t tid = pointsPtr[offset + i].tID;
		uint32_t time = pointsPtr[offset + i].time;
		if (x <= xmax &&x >= xmin&&y <= ymax&&y >= ymin) {
			resultPtrCuda[resultOffset[cellNo] + i].x = x;
			resultPtrCuda[resultOffset[cellNo] + i].y = y;
			resultPtrCuda[resultOffset[cellNo] + i].tID = tid;
			resultPtrCuda[resultOffset[cellNo] + i].time = time;
		}
		else
			resultPtrCuda[resultOffset[cellNo] + i].tID = -1;
	}
}

__global__ void cudaRangeQueryTest(RangeQueryStateTable* stateTable, int stateTableLength, uint8_t* result, 
	const int maxTrajNum) {
	int bID = blockIdx.x;
	int tID = threadIdx.x;

	__shared__ RangeQueryStateTable sharedStateTable;
	// __shared__ uint8_t resultTemp[10000]; //10K
	
	if(tID == 0)
		sharedStateTable = (stateTable[bID]); // ���岻�� �����������Ƴ����ڴ���

	__syncthreads();//block��threadͬ�� 

	/*
// 4+4*7=32byte
typedef struct RangeQueryStateTable {//  leafnode

	// GPU���
	void* ptr;	// ָ��GPU�ڴ�node��ָ�� �����洢
	int candidatePointNum;	// �����leafnode�еĽڵ���

	float xmin;
	float ymin;
	float xmax;
	float ymax;

	// cpu���
	int queryID;
	int startIdxInAllPoints; //startId in AllPoints����

}RangeQueryStateTable;*/

	int jobID = sharedStateTable.queryID;

	SPoint *baseAddr = (SPoint*)(sharedStateTable.ptr);
	int candidateNum = sharedStateTable.candidatePointNum;

	//int resultOffset = bID*maxPointNumInStateTable; 
	SPoint p;
	/*
	for (int i = 0; i <= candidateNum / MAXTHREAD-1; i++) {
		//p = *(baseAddr + (i*MAXTHREAD + tID));
		p = baseAddr[i*MAXTHREAD + tID];
		//result[i*MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
			//(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(i*MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
		//result[i*MAXTHREAD + tID + resultOffset].jobID = bID;
		//result[resultOffset + (i*MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && 
		//		(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
		if((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
			result[jobID*maxTrajNum + p.tID] = 1;
		//�����֤ͨ��������Ӧλ����Ϊ1

		//__syncthreads();
	}
	if (tID < candidateNum - candidateNum / MAXTHREAD * MAXTHREAD) {
		p = *(baseAddr + (candidateNum / MAXTHREAD * MAXTHREAD + tID));
		//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
		//	(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(candidateNum / MAXTHREAD * MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
		//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].jobID = bID;
		//result[resultOffset + (candidateNum / MAXTHREAD * MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) &&
		//	(p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
		if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
			result[jobID*maxTrajNum + p.tID] = 1;
	}
	*/

	/*
	example1:
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add<<<numBlocks, blockSize>>>(N, x, y);
	__global__
	void add(int n, float *x, float *y)
	{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x; // ��������stride��ͬ��
	for (int i = index; i < n; i += stride)
	y[i] = x[i] + y[i];
	}

	example2:
	add<<<1, 256>>>(N, x, y);
	__global__
	void add(int n, float *x, float *y)
	{
	  int index = threadIdx.x;
	  int stride = blockDim.x;
	  for (int i = index; i < n; i += stride)
		  y[i] = x[i] + y[i];
	}
	*/

	//  a new version
	// grid-stride loop��ʽ
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//int stride = blockDim.x * gridDim.x;
	//for (int i = index; i < n; i += stride) {
	//

	//}

	// stride = MAXTHREAD(blockDim.x)

	// kernel ��ÿ��thread �����첽ִ��
	//new version

	//for (int i = 0; i < candidateNum; i+=MAXTHREAD) {
	//	//p = *(baseAddr + (i*MAXTHREAD + tID));
	//	if (i + tID < candidateNum) {
	//		p = baseAddr[i + tID]; // �����ȫ���ڴ�ȡ��
	//					// p�Ǵ�ȫ���ڴ�ȡ��
	//		//result[i*MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
	//		//(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(i*MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
	//		//result[i*MAXTHREAD + tID + resultOffset].jobID = bID;
	//		//result[resultOffset + (i*MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && 
	//		//		(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
	//		if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
	//			result[jobID*maxTrajNum + p.tID] = 1;
	//		//�����֤ͨ��������Ӧλ����Ϊ1
	//	}
	//	//__syncthreads();
	//}




	for (int i = 0; i+ tID < candidateNum; i += MAXTHREAD) {
			//p = *(baseAddr + (i*MAXTHREAD + tID));

			p = baseAddr[i + tID];	// fetch from global memory

			if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
				result[jobID*maxTrajNum + p.tID] = 1; // write into global memory
			//jobID*maxTrajNum + p.tID jobID=queryID 1-40 maxTrajNum = this->trajNum + 1  p.tID is not thread, is trajID
			// basic idea
	}


	return;
	


	//else {
	//	//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = 0; //������Ĳ��֣�ֱ����Ϊ��Ч����
	//}
	//__syncthreads();
	//__syncthreads();
	//int globalTID = blockDim.x * blockIdx.x + threadIdx.x;
	//if (globalTID < stateTableLength) {

	//}
}

// offsetNum: how many uncontinuous part in each block
// offsetLen: how many elements in this uncontinuous part
// offset: the offset of all uncontinuous parts
// offsetInoffset: the offset in offset array for each block
__global__ void cudaRangeQueryTestWithoutMorton(RangeQueryStateTable* stateTable, int stateTableLength, uint8_t* result,
	const int maxTrajNum, int* offset, int* offsetLen, int* offsetNum, int* offsetInOffset) {
	int bID = blockIdx.x;
	int tID = threadIdx.x;
	__shared__ RangeQueryStateTable sharedStateTable;
	// __shared__ uint8_t resultTemp[10000]; //10K
	if (tID == 0)
		sharedStateTable = (stateTable[bID]);
	__syncthreads();
	int jobID = sharedStateTable.queryID;
	SPoint *baseAddr = (SPoint*)(sharedStateTable.ptr);
	int candidateNum = sharedStateTable.candidatePointNum;//��block����Ҫ��ѯ�ĵ�ĸ���
														  //int resultOffset = bID*maxPointNumInStateTable; //��block�Ľ������ʼ��ַ
	SPoint p;
	// all offset of start of array in this block
	__shared__ int offsetLocal[1000];
	__shared__ int offsetLenLocal[1000];
	int continuousNum = offsetNum[bID];
	for (int i = 0; i < continuousNum; i += MAXTHREAD) {
		if (i + tID < continuousNum) {
			offsetLocal[i + tID] = offset[offsetInOffset[bID] + i + tID];
			offsetLenLocal[i + tID] = offsetLen[offsetInOffset[bID] + i + tID];
		}
	}
	__syncthreads();
	/*
	for (int i = 0; i <= candidateNum / MAXTHREAD-1; i++) {
	//p = *(baseAddr + (i*MAXTHREAD + tID));
	p = baseAddr[i*MAXTHREAD + tID];
	//result[i*MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
	//(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(i*MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
	//result[i*MAXTHREAD + tID + resultOffset].jobID = bID;
	//result[resultOffset + (i*MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
	//		(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
	if((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
	result[jobID*maxTrajNum + p.tID] = 1;
	//�����֤ͨ��������Ӧλ����Ϊ1

	//__syncthreads();
	}
	if (tID < candidateNum - candidateNum / MAXTHREAD * MAXTHREAD) {
	p = *(baseAddr + (candidateNum / MAXTHREAD * MAXTHREAD + tID));
	//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
	//	(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(candidateNum / MAXTHREAD * MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
	//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].jobID = bID;
	//result[resultOffset + (candidateNum / MAXTHREAD * MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) &&
	//	(p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
	if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
	result[jobID*maxTrajNum + p.tID] = 1;
	}
	*/

	//new version
	for (int i = 0; i < continuousNum; i++) {
		int offsetLength = offsetLenLocal[i];
		int offsetAddr = offsetLocal[i];
		for (int j = 0; j < offsetLength; j += MAXTHREAD) {
			if (j + tID < offsetLength) {
				p = baseAddr[offsetAddr + j + tID];
				//result[i*MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
				//(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(i*MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
				//result[i*MAXTHREAD + tID + resultOffset].jobID = bID;
				//result[resultOffset + (i*MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && 
				//		(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
				if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
					result[jobID*maxTrajNum + p.tID] = 1;
				//�����֤ͨ��������Ӧλ����Ϊ1
			}
		}
	}
	return;



	//else {
	//	//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = 0; //������Ĳ��֣�ֱ����Ϊ��Ч����
	//}
	//__syncthreads();
	//__syncthreads();
	//int globalTID = blockDim.x * blockIdx.x + threadIdx.x;
	//if (globalTID < stateTableLength) {

	//}
}

__global__ void cudaRangeQuerySTIG(RangeQueryStateTable* stateTable, int stateTableLength, uint8_t* result,
	const int maxTrajNum) {
	int bID = blockIdx.x;
	int tID = threadIdx.x;
	//if (bID > stateTableLength)
	//	return;
	__shared__ RangeQueryStateTable sharedStateTable;
	// __shared__ uint8_t resultTemp[10000]; //10K
	if (tID == 0)
		sharedStateTable = (stateTable[bID]);
	__syncthreads();
	int jobID = sharedStateTable.queryID;
	SPoint *baseAddr = (SPoint*)(sharedStateTable.ptr);
	int candidateNum = sharedStateTable.candidatePointNum;//��block����Ҫ��ѯ�ĵ�ĸ���
														  //int resultOffset = bID*maxPointNumInStateTable; //��block�Ľ������ʼ��ַ
	SPoint p;
	//new version
	for (int i = 0; i < candidateNum; i += MAXTHREAD) {
		//p = *(baseAddr + (i*MAXTHREAD + tID));
		if (i + tID < candidateNum) {
			p = baseAddr[i + tID];
			//result[i*MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
			//(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(i*MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
			//result[i*MAXTHREAD + tID + resultOffset].jobID = bID;
			//result[resultOffset + (i*MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && 
			//		(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
			if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
				result[jobID*maxTrajNum + p.tID] = 1;
			//�����֤ͨ��������Ӧλ����Ϊ1
		}
		//__syncthreads();
	}



	//else {
	//	//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = 0; //������Ĳ��֣�ֱ����Ϊ��Ч����
	//}
	//__syncthreads();
	//__syncthreads();
	//int globalTID = blockDim.x * blockIdx.x + threadIdx.x;
	//if (globalTID < stateTableLength) {

	//}
}

int cudaRangeQuerySTIGHandler(RangeQueryStateTable* stateTableGPU, int stateTableLength, uint8_t *result, int maxTrajNum
	, int maxQueryNum, hipStream_t stream)
{
	//RangeQueryResultGPU* resultGPU;
	//MyTimer timer;
	uint8_t* resultGPU;
	//int resultByteNum = (maxPointNum)/8+1;//ÿ�������Ҫ�ü���byte���棬���ܰ����أ�ֻ�ܰ��ֽ�
	CUDA_CALL(hipMalloc((void**)&resultGPU, (maxTrajNum)*maxQueryNum));//selective��һ��
	CUDA_CALL(hipMemset(resultGPU, 0, (maxTrajNum)*maxQueryNum));
	//timer.start();
	//�����һ���ڴ棬ÿ��stateTable��ռ�ݵ��ڴ������
	//CUDA_CALL(hipMalloc((void**)&resultGPU, (maxPointNum)*stateTableLength));

	//CUDA_CALL(hipMalloc((void**)&resultGPU, maxPointNum*stateTableLength*sizeof(RangeQueryResultGPU)));
	//timer.stop();
	//std::cout << "Time 1:" << timer.elapse() << "ms" << std::endl;

	//timer.start();	
	cudaRangeQuerySTIG << <stateTableLength, MAXTHREAD, 0, stream >> >(stateTableGPU, stateTableLength, resultGPU, maxTrajNum);
	CUDA_CALL(hipDeviceSynchronize());
	//timer.stop();
	//std::cout << "Time 2:" << timer.elapse() << "ms" << std::endl;

	//timer.start();

	CUDA_CALL(hipMemcpy(result, resultGPU, (maxTrajNum)*maxQueryNum, hipMemcpyDeviceToHost));

	//timer.stop();
	//std::cout << "Time 3:" << timer.elapse() << "ms" << std::endl;
	CUDA_CALL(hipFree(resultGPU));
	return 0;
}

int cudaRangeQueryTestHandlerNonMorton(RangeQueryStateTable* stateTableGPU, int stateTableLength, uint8_t *result, int maxTrajNum
	, int maxJobNum, hipStream_t stream, int* offset, int* offsetLen, int* offsetNum, int* offsetInOffset) {
	/*
	without Morton encoding, each line should be processed seperately.
	divide loops to process each line (continuous points).

	*/
	//RangeQueryResultGPU* resultGPU;
	//MyTimer timer;
	uint8_t* resultGPU;
	//int resultByteNum = (maxPointNum)/8+1;//ÿ�������Ҫ�ü���byte���棬���ܰ����أ�ֻ�ܰ��ֽ�

	CUDA_CALL(hipMalloc((void**)&resultGPU, (maxTrajNum)*maxJobNum));//selective��һ��
	CUDA_CALL(hipMemset(resultGPU, 0, (maxTrajNum)*maxJobNum));
	
	//timer.start();
	//�����һ���ڴ棬ÿ��stateTable��ռ�ݵ��ڴ������
	//CUDA_CALL(hipMalloc((void**)&resultGPU, (maxPointNum)*stateTableLength));

	//CUDA_CALL(hipMalloc((void**)&resultGPU, maxPointNum*stateTableLength*sizeof(RangeQueryResultGPU)));
	//timer.stop();
	//std::cout << "Time 1:" << timer.elapse() << "ms" << std::endl;

	//timer.start();	
	cudaRangeQueryTestWithoutMorton << <stateTableLength, MAXTHREAD, 0, stream >> >(stateTableGPU, stateTableLength, 
		resultGPU, maxTrajNum, offset, offsetLen, offsetNum, offsetInOffset);
	CUDA_CALL(hipDeviceSynchronize());
	//timer.stop();
	//std::cout << "Time 2:" << timer.elapse() << "ms" << std::endl;
	//timer.start();

	CUDA_CALL(hipMemcpy(result, resultGPU, (maxTrajNum)*maxJobNum, hipMemcpyDeviceToHost));
	CUDA_CALL(hipFree(resultGPU));

	//timer.stop();
	//std::cout << "Time 3:" << timer.elapse() << "ms" << std::endl;
	return 0;
}

// cudaRangeQueryTestHandler((RangeQueryStateTable*)this->stateTableGPU[device_idx], this->stateTableLength[device_idx], resultsReturned, this->trajNum + 1, rangeNum, stream);

int cudaRangeQueryTestHandler(RangeQueryStateTable* stateTableGPU, int stateTableLength, uint8_t *result, int maxTrajNum
	, int maxJobNum, hipStream_t stream) {
	//RangeQueryResultGPU* resultGPU;
	//MyTimer timer;

	uint8_t* resultGPU; // ָ��GPU
	//int resultByteNum = (maxPointNum)/8+1;//ÿ�������Ҫ�ü���byte���棬���ܰ����أ�ֻ�ܰ��ֽ�

	CUDA_CALL(hipMalloc((void**)&resultGPU, (maxTrajNum)*maxJobNum));//selective��һ��
	CUDA_CALL(hipMemset(resultGPU, 0, (maxTrajNum)*maxJobNum));

	//timer.start();
	//�����һ���ڴ棬ÿ��stateTable��ռ�ݵ��ڴ������
	//CUDA_CALL(hipMalloc((void**)&resultGPU, (maxPointNum)*stateTableLength));
	
	//CUDA_CALL(hipMalloc((void**)&resultGPU, maxPointNum*stateTableLength*sizeof(RangeQueryResultGPU)));
	//timer.stop();
	//std::cout << "Time 1:" << timer.elapse() << "ms" << std::endl;
	
	//timer.start();

	//����stateTableLength��filteringʱ���壬1 block ��Ӧһ�� quad-fake-leaf-node 
	cudaRangeQueryTest <<< stateTableLength, MAXTHREAD,0, stream >>>(stateTableGPU, stateTableLength, resultGPU, maxTrajNum);
	
	// // Wait for GPU to finish before accessing on host ��ʽͬ���������õ��� resultGPU

	CUDA_CALL(hipDeviceSynchronize());//2GPUҲ����ͨ�ţ��� ������


	// CUDA_CALL(hipDeviceSynchronize(stream)); // Ϊʲôû��stream

	//timer.stop();
	//std::cout << "Time 2:" << timer.elapse() << "ms" << std::endl;
	//timer.start();
	
	// GPU->CPU ���ؽ��
	CUDA_CALL(hipMemcpyAsync(result, resultGPU, (maxTrajNum)*maxJobNum, hipMemcpyDeviceToHost));
	//CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipFree(resultGPU)); // not recommended

	//timer.stop();
	//std::cout << "Time 3:" << timer.elapse() << "ms" << std::endl;
	return 0;
}

/*
int cudaRangeQueryHandler(int* candidateCells, int* rangeStarts, int* rangeEnds, int candidateCellNum,float xmin, float ymin, float xmax, float ymax, Point*& resultsGPU, int& resultNum,Point *pointsPtrGPU,Point *&result) {
	//��һ��������ʱû���ã�ע������candidatecells[i]�Ѿ�������cell��id������ֻ��ǿ�
	//���ĸ�������ʾ�ǿյ�cell����
	//����candidate���еĵ��������gpu�ڿ�����ͬ��С�Ŀռ���flag��rangestart��rangeend����Ӧcandidatecell�ڵĲ�������AllPoints����ʼ�±����ֹ�±�
	//�����������͵����ڶ��������������һ���Ǳ�������GPU��ַ���ڶ����ǽ���ĸ���
	//PointsPtrGPU�����ݼ���gpu�ĵ�ַ
	MyTimer timer1;
	timer1.start();
	int counter = 0;
	int *resultOffset = (int*)malloc(sizeof(int)*candidateCellNum);
	//std::cout << candidateCellNum << ":"<<std::endl;
	for (int i = 0; i <= candidateCellNum - 1; i++) {
		resultOffset[i] = counter;
		////debug
		//std::cout << "(" << rangeStarts[i] << "," << rangeEnds[i] << ");"<<"["<<resultOffset[i]<<"]";
		////debug
		counter += rangeEnds[i] - rangeStarts[i] + 1;
	}
	int totalPointNumInCandidate = counter;


	int *rangeStartsCuda = NULL, *rangeEndsCuda = NULL, *resultOffsetCuda = NULL;

	CUDA_CALL(hipMalloc((void**)&resultsGPU, sizeof(Point)*totalPointNumInCandidate));
	//��range��cell��Ϣд��gpu
	//CUDA_CALL(hipMalloc((void**)&candidateCellsCuda, sizeof(int)*candidateCellNum));
	CUDA_CALL(hipMalloc((void**)&rangeStartsCuda, candidateCellNum*sizeof(int)));
	//std::cout << "\n" << candidateCellNum*sizeof(int) << "\n";
	CUDA_CALL(hipMalloc((void**)&rangeEndsCuda, candidateCellNum*sizeof(int)));
	CUDA_CALL(hipMalloc((void**)&resultOffsetCuda, candidateCellNum*sizeof(int)));
	//CUDA_CALL(hipMemcpy(candidateCellsCuda, candidateCells, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(rangeStartsCuda, rangeStarts, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpy(rangeEndsCuda, rangeEnds, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(resultOffsetCuda, resultOffset, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	////debug
	//CUDA_CALL(hipMemcpy( rangeStarts, rangeStartsCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CALL(hipMemcpy( rangeEnds, rangeEndsCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CALL(hipMemcpy( resultOffset, resultOffsetCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i <= candidateCellNum - 1; i++) {
	//	//debug
	//	std::cout << "(" << rangeStarts[i] << "," << rangeEnds[i] << ");" << "[" << resultOffset[i] << "]";
	//	//debug
	//}
	//debug
	timer1.stop();
	std::cout << timer1.ticks() << std::endl;
	timer1.start();
	//����kernel�����ĳ����������������Ӧλ��д����AllPoints�е��±꣬����д��-1
	//ÿ��cell�����һ��block
	cudaRangeQuery <<<candidateCellNum, MAXTHREAD >>>(rangeStartsCuda, rangeEndsCuda, candidateCellNum, pointsPtrGPU, xmin, ymin, xmax, ymax, resultOffsetCuda, resultsGPU);
	//kernel���ý��������������idxsGPU�У����������������ӦԪ����������AllPoints���±꣬��������ϣ�����Ϊ-1
	//CUDA_CALL(hipFree(candidateCellsCuda));
	CUDA_CALL(hipFree(rangeStartsCuda));
	CUDA_CALL(hipFree(rangeEndsCuda));
	CUDA_CALL(hipFree(resultOffsetCuda));
	//getLastCudaError("Error in Calling 'kernel'");
	//ʹ��Thrustɾ������-1���õ����ս��
	timer1.stop();
	std::cout << timer1.ticks() << std::endl;

	//���н���ϲ�
	//test

	timer1.start();
	Point *resultset = NULL;
	resultset = (Point*)malloc(totalPointNumInCandidate*sizeof(Point));
	CUDA_CALL(hipMemcpy(resultset, resultsGPU, sizeof(Point)*totalPointNumInCandidate, hipMemcpyDeviceToHost));
	std::vector<Point> *resultPoint = new std::vector<Point>;
	for (int i = 0; i <= totalPointNumInCandidate - 1; i++) {
		if (resultset[i].tID != -1)
		{
			resultPoint->push_back(resultset[i]);
		}
	}
	result = &resultPoint->at(0);
	free(resultset);
	//test
	timer1.stop();
	std::cout << timer1.ticks() << std::endl;
	
	//���н���ϲ�
	
	//thrust::device_ptr<int> idxsPtr = thrust::device_pointer_cast(idxsGPU);
	//int a;
	//hipMemcpy(&a, idxsGPU, 1, hipMemcpyDeviceToHost);
	//size_t num = thrust::remove(idxsPtr, idxsPtr + totalPointNumInCandidate-1, -1) - idxsPtr;
	//int *result = (int*)malloc(sizeof(int)*num);
	//thrust::copy(idxsPtr, idxsPtr + num, result);
	//resultNum = num;
	//resultIdx = result;

	//CUDA_CALL(hipFree(idxsGPU));


	return 0;
}
*/


#else
int cudaRangeQueryHandler(Point* pointsPtr, int pointNum, float xmin, float ymin, float xmax, float ymax, Point*& resultsPtr, int& resultNum) {
	Point* pointsPtrCuda = NULL;
	Point* resultPtrCuda = NULL;
	CUDA_CALL(hipMalloc((void**)&pointsPtrCuda, pointNum * sizeof(Point))); //�������ݵ��ڴ�
	CUDA_CALL(hipMalloc((void**)&resultPtrCuda, pointNum * sizeof(Point))); //gpu�ڴ洢����ĵط�
	CUDA_CALL(hipMemcpy(pointsPtrCuda, pointsPtr, pointNum * sizeof(Point), hipMemcpyHostToDevice));//���ݿ�����gpu��

																									  //���ú˺����������ݣ��������gpu��

																									  //ȡ�����ݣ�����
	return 0;
}
#endif




//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
// Helper function for using CUDA to add vectors in parallel.
